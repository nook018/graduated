#include "hip/hip_runtime.h"
#include	<stdio.h>
#include	<string.h>
#include	"err_handler.h"
#include	"filter_db.h"
#include	"util.h"
#include    <sys/time.h>
#include    <fstream>
#include    <iostream>

#include    <hip/hip_runtime.h>

#define		BUF_SZ   256
using namespace std;
clsFilterDB   db;

#define cudaCheckErrors(msg) \
    do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                        msg, hipGetErrorString(__err), __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
    } while (0)

inline __host__ __device__ UINT64 genPrefix(UINT64 prefix, unsigned char len)
{
    UINT64 result;
    UINT mask_m;
    
    if (len == 0)
	return(0);
    else{
	// mask    = 0xFFFFFFFFFFFFFFFFLL << (64-len);
	mask_m    = 0xFFFFFFFF << (32-len);
        result  = prefix & mask_m;
        return(result);
    }
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec*1e6 + (double)tp.tv_usec);
}


void bubble_sort(clsFilterDB item, int len)
{
    int i, j;
    filter temp;
    for(i = 0; i < len - 1; i++)
        for(j = 0; j < len - i - 1; j++)
        {
            if(item.filterSet_[j].destIPLen_ > item.filterSet_[j+1].destIPLen_)
            {
                
                //printf("1.srcIP: %d\n", item.filterSet_[j].srcIPLen_);
                temp = item.filterSet_[j];
                item.filterSet_[j] = item.filterSet_[j+1];
                item.filterSet_[j+1] = temp;
                //printf("2.srcIP: %d\n", item.filterSet_[j].srcIPLen_);
            }
            if(item.filterSet_[j].srcIPLen_ > item.filterSet_[j+1].srcIPLen_)
            {
                //printf("1.srcIP: %d\n", item.filterSet_[j].srcIPLen_);
                temp = item.filterSet_[j];
                item.filterSet_[j] = item.filterSet_[j+1];
                item.filterSet_[j+1] = temp;
                //printf("2.srcIP: %d\n", item.filterSet_[j].srcIPLen_);
            }
            
        }
    
}

// distribute filters before sorting, to thread by order. (approach 1)
__global__ void gpu_conflict_detect_0(filter * __restrict__ filterSet, int rule_size, dev_var* devTotal)
{
    //printf("blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x, threadIdx.x);
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int threadSize = gridDim.x * blockDim.x;
    int base = rule_size/threadSize+1;
    int start = threadID*base;
    //devTotal[start].set = 1;
    //devTotal[threadID].set = 1;
    //devTotal[threadID].result = 0;
    //printf("threadSize:%d,%d\n", threadSize, rule_size/threadSize + 1);
    //for(int i=threadID; i<rule_size; i+=threadSize){  
    for (int i=start; i<(start+base) && i<rule_size ; i++) {
	devTotal[i].set = 1;        
	devTotal[i].result = 0;
        for (int j= 0; j<rule_size ; j++) {
            if( filterSet[i].srcIPLen_ > filterSet[j].srcIPLen_ ){
                if( genPrefix(filterSet[i].srcIP_, filterSet[j].srcIPLen_) == filterSet[j].srcIP_ ){
                    if(( (!filterSet[i].pro_num_ || !filterSet[j].pro_num_) || (filterSet[i].pro_num_ == filterSet[j].pro_num_ )) ) {
                        if (
                            (
                             
                             ((filterSet[i].destPortleft_ <= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ >= filterSet[j].destPortleft_))   ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ <= filterSet[j].destPortright_)) ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_))
                             
                             )
                            &&
                            (
                             ((filterSet[i].srcPortleft_ <= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ >= filterSet[j].srcPortleft_))   ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortleft_) && (filterSet[i].srcPortright_ <= filterSet[j].srcPortright_)) ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ <= filterSet[j].srcPortleft_))
                             
                             )
                            )
                        {
                            
                            
                            if (filterSet[i].destIPLen_ > filterSet[j].destIPLen_ ){
                                if ( genPrefix(filterSet[i].destIP_, filterSet[j].destIPLen_) == filterSet[j].destIP_){
                                    //counter1++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        devTotal[i].result++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ < filterSet[j].destIPLen_){
                                if ( genPrefix(filterSet[j].destIP_, filterSet[i].destIPLen_) == filterSet[i].destIP_){
                                    //counter2++;
                                    /*
                                     if(
                                     ( (rule.filterSet_[i].destPortleft_ <= rule.filterSet_[j].destPortleft_) && (rule.filterSet_[i].destPortright_ >= rule.filterSet_[j].destPortright_) ) &&
                                     ( (rule.filterSet_[i].srcPortleft_  <= rule.filterSet_[j].srcPortleft_)  && (rule.filterSet_[i].srcPortright_  >= rule.filterSet_[j].srcPortright_) ) &&
                                     ( (rule.filterSet_[i].pro_num_ == rule.filterSet_[j].pro_num_) || (rule.filterSet_[i].pro_num_ == 0) )
                                     ){}
                                     else
                                     */
                                    //devTotal++;
                                    devTotal[i].result++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ == filterSet[j].destIPLen_){
                                if ( filterSet[i].destIP_ == filterSet[j].destIP_){
                                    //counter3++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        devTotal[i].result++;
                                }
                            }
                        }
                    }
                }
            }
        }
    }
    
}


// base on new, distribute filters to thread by order. (approach 1)
__global__ void gpu_conflict_detect_1(filter * __restrict__ filterSet, int rule_size, dev_var* devTotal)
{
    //printf("blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x, threadIdx.x);
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int threadSize = gridDim.x * blockDim.x;
    int base = rule_size/threadSize+1;
    int start = threadID*base;
    //devTotal[start].set = 1;
    //devTotal[threadID].set = 1;
    //devTotal[threadID].result = 0;
    //printf("threadSize:%d,%d\n", threadSize, rule_size/threadSize + 1);
    //for(int i=threadID; i<rule_size; i+=threadSize){  
    for (int i=start; i<(start+base) && i<rule_size ; i++) {
	devTotal[i].set = 1;        
	devTotal[i].result = 0;
        for (int j= 0; j<i ; j++) {
            if( filterSet[i].srcIPLen_ > filterSet[j].srcIPLen_ ){
                if( genPrefix(filterSet[i].srcIP_, filterSet[j].srcIPLen_) == filterSet[j].srcIP_ ){
                    if(( (!filterSet[i].pro_num_ || !filterSet[j].pro_num_) || (filterSet[i].pro_num_ == filterSet[j].pro_num_ )) ) {
                        if (
                            (
                             
                             ((filterSet[i].destPortleft_ <= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ >= filterSet[j].destPortleft_))   ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ <= filterSet[j].destPortright_)) ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_))
                             
                             )
                            &&
                            (
                             ((filterSet[i].srcPortleft_ <= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ >= filterSet[j].srcPortleft_))   ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortleft_) && (filterSet[i].srcPortright_ <= filterSet[j].srcPortright_)) ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ <= filterSet[j].srcPortleft_))
                             
                             )
                            )
                        {
                            
                            
                            if (filterSet[i].destIPLen_ > filterSet[j].destIPLen_ ){
                                if ( genPrefix(filterSet[i].destIP_, filterSet[j].destIPLen_) == filterSet[j].destIP_){
                                    //counter1++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        devTotal[i].result++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ < filterSet[j].destIPLen_){
                                if ( genPrefix(filterSet[j].destIP_, filterSet[i].destIPLen_) == filterSet[i].destIP_){
                                    //counter2++;
                                    /*
                                     if(
                                     ( (rule.filterSet_[i].destPortleft_ <= rule.filterSet_[j].destPortleft_) && (rule.filterSet_[i].destPortright_ >= rule.filterSet_[j].destPortright_) ) &&
                                     ( (rule.filterSet_[i].srcPortleft_  <= rule.filterSet_[j].srcPortleft_)  && (rule.filterSet_[i].srcPortright_  >= rule.filterSet_[j].srcPortright_) ) &&
                                     ( (rule.filterSet_[i].pro_num_ == rule.filterSet_[j].pro_num_) || (rule.filterSet_[i].pro_num_ == 0) )
                                     ){}
                                     else
                                     */
                                    //devTotal++;
                                    devTotal[i].result++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ == filterSet[j].destIPLen_){
                                if ( filterSet[i].destIP_ == filterSet[j].destIP_){
                                    //counter3++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        devTotal[i].result++;
                                }
                            }
                        }
                    }
                }
            }
        }
    }
    
}

// base on new, distribute filters to thread by work. (approach 2)
__global__ void gpu_conflict_detect_2(filter * __restrict__ filterSet, int rule_size, dev_var* devTotal)
{
    //printf("blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x, threadIdx.x);
    int threadID = threadIdx.x * gridDim.x + blockIdx.x;
    int threadSize = gridDim.x * blockDim.x;
    
    //devTotal[threadID].set = 1;
    //printf("threadSize:%d,%d\n", threadSize, rule_size/threadSize + 1);
    for (int i=threadID; i<rule_size; i+=threadSize) {
    //for(int i=threadID; i<rule_size; i+=threadSize){  
        devTotal[i].set = 1;
        //devTotal[i].result = 0;
        for (int j= 0; j<i ; j++) {
            if( filterSet[i].srcIPLen_ > filterSet[j].srcIPLen_ ){
                if( genPrefix(filterSet[i].srcIP_, filterSet[j].srcIPLen_) == filterSet[j].srcIP_ ){
                    if(( (!filterSet[i].pro_num_ || !filterSet[j].pro_num_) || (filterSet[i].pro_num_ == filterSet[j].pro_num_ )) ) {
                        if (
                            (
                             
                             ((filterSet[i].destPortleft_ <= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ >= filterSet[j].destPortleft_))   ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ <= filterSet[j].destPortright_)) ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_))
                             
                             )
                            &&
                            (
                             ((filterSet[i].srcPortleft_ <= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ >= filterSet[j].srcPortleft_))   ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortleft_) && (filterSet[i].srcPortright_ <= filterSet[j].srcPortright_)) ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ <= filterSet[j].srcPortleft_))
                             
                             )
                            )
                        {
                            
                            
                            if (filterSet[i].destIPLen_ > filterSet[j].destIPLen_ ){
                                if ( genPrefix(filterSet[i].destIP_, filterSet[j].destIPLen_) == filterSet[j].destIP_){
                                    //counter1++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        devTotal[i].result++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ < filterSet[j].destIPLen_){
                                if ( genPrefix(filterSet[j].destIP_, filterSet[i].destIPLen_) == filterSet[i].destIP_){
                                    //counter2++;
                                    /*
                                     if(
                                     ( (rule.filterSet_[i].destPortleft_ <= rule.filterSet_[j].destPortleft_) && (rule.filterSet_[i].destPortright_ >= rule.filterSet_[j].destPortright_) ) &&
                                     ( (rule.filterSet_[i].srcPortleft_  <= rule.filterSet_[j].srcPortleft_)  && (rule.filterSet_[i].srcPortright_  >= rule.filterSet_[j].srcPortright_) ) &&
                                     ( (rule.filterSet_[i].pro_num_ == rule.filterSet_[j].pro_num_) || (rule.filterSet_[i].pro_num_ == 0) )
                                     ){}
                                     else
                                     */
                                    //devTotal++;
                                    devTotal[i].result++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ == filterSet[j].destIPLen_){
                                if ( filterSet[i].destIP_ == filterSet[j].destIP_){
                                    //counter3++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        devTotal[i].result++;
                                }
                            }
                        }
                    }
                }
            }
        }
    }
    
}

// base on new, distribute filters to thread by work. (compare)
__global__ void gpu_conflict_detect_3(filter * __restrict__ filterSet, int rule_size, dev_var* devTotal)
{
    //printf("blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x, threadIdx.x);
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int threadSize = gridDim.x * blockDim.x;
    //int base = rule_size/threadSize+1;
    //int start = threadID*base;
    //devTotal[start].set = 1;
    //devTotal[start].result = 0;
    //printf("threadSize:%d,%d\n", threadSize, rule_size/threadSize + 1);
    for (int i=threadID; i<rule_size; i+=threadSize) {
        devTotal[i].set = 1;
	//devTotal[i].result = 0;
        for (int j= 0; j<i ; j++) {
            if( filterSet[i].srcIPLen_ > filterSet[j].srcIPLen_ ){
                if( genPrefix(filterSet[i].srcIP_, filterSet[j].srcIPLen_) == filterSet[j].srcIP_ ){
                    if(( (!filterSet[i].pro_num_ || !filterSet[j].pro_num_) || (filterSet[i].pro_num_ == filterSet[j].pro_num_ )) ) {
                        if (
                            (
                             
                             ((filterSet[i].destPortleft_ <= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ >= filterSet[j].destPortleft_))   ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ <= filterSet[j].destPortright_)) ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_))
                             
                             )
                            &&
                            (
                             ((filterSet[i].srcPortleft_ <= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ >= filterSet[j].srcPortleft_))   ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortleft_) && (filterSet[i].srcPortright_ <= filterSet[j].srcPortright_)) ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ <= filterSet[j].srcPortleft_))
                             
                             )
                            )
                        {
                            
                            
                            if (filterSet[i].destIPLen_ > filterSet[j].destIPLen_ ){
                                if ( genPrefix(filterSet[i].destIP_, filterSet[j].destIPLen_) == filterSet[j].destIP_){
                                    //counter1++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        devTotal[i].result++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ < filterSet[j].destIPLen_){
                                if ( genPrefix(filterSet[j].destIP_, filterSet[i].destIPLen_) == filterSet[i].destIP_){
                                    //counter2++;
                                    /*
                                     if(
                                     ( (rule.filterSet_[i].destPortleft_ <= rule.filterSet_[j].destPortleft_) && (rule.filterSet_[i].destPortright_ >= rule.filterSet_[j].destPortright_) ) &&
                                     ( (rule.filterSet_[i].srcPortleft_  <= rule.filterSet_[j].srcPortleft_)  && (rule.filterSet_[i].srcPortright_  >= rule.filterSet_[j].srcPortright_) ) &&
                                     ( (rule.filterSet_[i].pro_num_ == rule.filterSet_[j].pro_num_) || (rule.filterSet_[i].pro_num_ == 0) )
                                     ){}
                                     else
                                     */
                                    //devTotal++;
                                    devTotal[i].result++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ == filterSet[j].destIPLen_){
                                if ( filterSet[i].destIP_ == filterSet[j].destIP_){
                                    //counter3++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        devTotal[i].result++;
                                }
                            }
                        }
                    }
                }
            }
        }
    }
    
}

// base on new, distribute filters to thread by work. (1234 8765)
__global__ void gpu_conflict_detect_4(filter *filterSet, int rule_size, dev_var* devTotal)
{
    //printf("blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x, threadIdx.x);
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int threadSize = gridDim.x * blockDim.x;
    int base = threadSize * 2;
    int start = (base-1) - threadID;
    //devTotal[start].set = 1;
    //devTotal[start].result = 0;
    //printf("threadSize:%d,%d\n", threadSize, rule_size/threadSize + 1);
    for (int i=threadID; i<rule_size; i+=base) {
        devTotal[i].set = 1;
        devTotal[i].result = 0;
        for (int j= 0; j<i ; j++) {
            if( filterSet[i].srcIPLen_ > filterSet[j].srcIPLen_ ){
                if( genPrefix(filterSet[i].srcIP_, filterSet[j].srcIPLen_) == filterSet[j].srcIP_ ){
                    if(( (!filterSet[i].pro_num_ || !filterSet[j].pro_num_) || (filterSet[i].pro_num_ == filterSet[j].pro_num_ )) ) {
                        if (
                            (
                             
                             ((filterSet[i].destPortleft_ <= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ >= filterSet[j].destPortleft_))   ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ <= filterSet[j].destPortright_)) ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_))
                             
                             )
                            &&
                            (
                             ((filterSet[i].srcPortleft_ <= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ >= filterSet[j].srcPortleft_))   ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortleft_) && (filterSet[i].srcPortright_ <= filterSet[j].srcPortright_)) ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ <= filterSet[j].srcPortleft_))
                             
                             )
                            )
                        {
                            
                            
                            if (filterSet[i].destIPLen_ > filterSet[j].destIPLen_ ){
                                if ( genPrefix(filterSet[i].destIP_, filterSet[j].destIPLen_) == filterSet[j].destIP_){
                                    //counter1++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        devTotal[i].result++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ < filterSet[j].destIPLen_){
                                if ( genPrefix(filterSet[j].destIP_, filterSet[i].destIPLen_) == filterSet[i].destIP_){
                                    //counter2++;
                                    /*
                                     if(
                                     ( (rule.filterSet_[i].destPortleft_ <= rule.filterSet_[j].destPortleft_) && (rule.filterSet_[i].destPortright_ >= rule.filterSet_[j].destPortright_) ) &&
                                     ( (rule.filterSet_[i].srcPortleft_  <= rule.filterSet_[j].srcPortleft_)  && (rule.filterSet_[i].srcPortright_  >= rule.filterSet_[j].srcPortright_) ) &&
                                     ( (rule.filterSet_[i].pro_num_ == rule.filterSet_[j].pro_num_) || (rule.filterSet_[i].pro_num_ == 0) )
                                     ){}
                                     else
                                     */
                                    //devTotal++;
                                    devTotal[i].result++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ == filterSet[j].destIPLen_){
                                if ( filterSet[i].destIP_ == filterSet[j].destIP_){
                                    //counter3++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        devTotal[i].result++;
                                }
                            }
                        }
                    }
                }
            }
        }
    }
    
    for (int i=start; i<rule_size; i+=base) {
        devTotal[i].set = 1;
        devTotal[i].result = 0;
        for (int j= 0; j<i ; j++) {
            if( filterSet[i].srcIPLen_ > filterSet[j].srcIPLen_ ){
                if( genPrefix(filterSet[i].srcIP_, filterSet[j].srcIPLen_) == filterSet[j].srcIP_ ){
                    if(( (!filterSet[i].pro_num_ || !filterSet[j].pro_num_) || (filterSet[i].pro_num_ == filterSet[j].pro_num_ )) ) {
                        if (
                            (
                             
                             ((filterSet[i].destPortleft_ <= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ >= filterSet[j].destPortleft_))   ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ <= filterSet[j].destPortright_)) ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_))
                             
                             )
                            &&
                            (
                             ((filterSet[i].srcPortleft_ <= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ >= filterSet[j].srcPortleft_))   ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortleft_) && (filterSet[i].srcPortright_ <= filterSet[j].srcPortright_)) ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ <= filterSet[j].srcPortleft_))
                             
                             )
                            )
                        {
                            
                            
                            if (filterSet[i].destIPLen_ > filterSet[j].destIPLen_ ){
                                if ( genPrefix(filterSet[i].destIP_, filterSet[j].destIPLen_) == filterSet[j].destIP_){
                                    //counter1++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        devTotal[i].result++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ < filterSet[j].destIPLen_){
                                if ( genPrefix(filterSet[j].destIP_, filterSet[i].destIPLen_) == filterSet[i].destIP_){
                                    //counter2++;
                                    /*
                                     if(
                                     ( (rule.filterSet_[i].destPortleft_ <= rule.filterSet_[j].destPortleft_) && (rule.filterSet_[i].destPortright_ >= rule.filterSet_[j].destPortright_) ) &&
                                     ( (rule.filterSet_[i].srcPortleft_  <= rule.filterSet_[j].srcPortleft_)  && (rule.filterSet_[i].srcPortright_  >= rule.filterSet_[j].srcPortright_) ) &&
                                     ( (rule.filterSet_[i].pro_num_ == rule.filterSet_[j].pro_num_) || (rule.filterSet_[i].pro_num_ == 0) )
                                     ){}
                                     else
                                     */
                                    //devTotal++;
                                    devTotal[i].result++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ == filterSet[j].destIPLen_){
                                if ( filterSet[i].destIP_ == filterSet[j].destIP_){
                                    //counter3++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        devTotal[i].result++;
                                }
                            }
                        }
                    }
                }
            }
        }
    }

    
}

void host_conflict_detect(filter *filterSet, int rule_size, int &total)
{
    for (int i=0; i<rule_size; i++)	{
        for (int j= 0; j<i; j++) {
            if( (filterSet[i].srcIPLen_ > filterSet[j].srcIPLen_ ) && ( genPrefix(filterSet[i].srcIP_, filterSet[j].srcIPLen_) == filterSet[j].srcIP_ ) ){
                if(( (!filterSet[i].pro_num_ || !filterSet[j].pro_num_) || (filterSet[i].pro_num_ == filterSet[j].pro_num_ )) ) {
                    if (
                        (
                         
                         ((filterSet[i].destPortleft_ <= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ >= filterSet[j].destPortleft_))   ||
                         ((filterSet[i].destPortright_ >= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ <= filterSet[j].destPortright_)) ||
                         ((filterSet[i].destPortright_ >= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_))
                         
                        )
                        &&
                        (
                         ((filterSet[i].srcPortleft_ <= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ >= filterSet[j].srcPortleft_))   ||
                         ((filterSet[i].srcPortright_ >= filterSet[j].srcPortleft_) && (filterSet[i].srcPortright_ <= filterSet[j].srcPortright_)) ||
                         ((filterSet[i].srcPortright_ >= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ <= filterSet[j].srcPortleft_))
                         
                         )
                        )
                    {
                        
                        
                        if (filterSet[i].destIPLen_ > filterSet[j].destIPLen_ ){
                            if ( genPrefix(filterSet[i].destIP_, filterSet[j].destIPLen_) == filterSet[j].destIP_){
                                //counter1++;
                                
                                if(
                                   ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                   ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                   ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                   ){}
                                else
                                    total++;
                            }
                        }
                        
                        if (filterSet[i].destIPLen_ < filterSet[j].destIPLen_){
                            if ( genPrefix(filterSet[j].destIP_, filterSet[i].destIPLen_) == filterSet[i].destIP_){
                                //counter2++;
                                /*
                                 if(
                                 ( (rule.filterSet_[i].destPortleft_ <= rule.filterSet_[j].destPortleft_) && (rule.filterSet_[i].destPortright_ >= rule.filterSet_[j].destPortright_) ) &&
                                 ( (rule.filterSet_[i].srcPortleft_  <= rule.filterSet_[j].srcPortleft_)  && (rule.filterSet_[i].srcPortright_  >= rule.filterSet_[j].srcPortright_) ) &&
                                 ( (rule.filterSet_[i].pro_num_ == rule.filterSet_[j].pro_num_) || (rule.filterSet_[i].pro_num_ == 0) )
                                 ){}
                                 else
                                 */
                                total++;
                            }
                        }
                        
                        if (filterSet[i].destIPLen_ == filterSet[j].destIPLen_){
                            if ( filterSet[i].destIP_ == filterSet[j].destIP_){
                                //counter3++;
                                
                                if(
                                   ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                   ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                   ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                   ){}
                                else
                                    total++;
                            }
                        }
                        
                    }
                }
            }
        }
    }
    
}


int main(int argc, char *argv[])
{

    ofstream fout;
    double sort_start, hconflict_start, g1conflict_start, g2conflict_start, g0conflict_start;
    double sort_time, hconflict_time, g1conflict_time, g2conflict_time, g0conflict_time;
    int db_size;
    int total = 0;
    //int *cpy_dev_total;
    dev_var *cpy_dev;

    // CUDA variable
    filter* dev_filterSet;	// store filter in device
    //int dev_total = 0;
    dev_var* dev_total;		// store conflict result in device
    
    //insert filters
    if (db.loadFilters(argv[1]) == -1)
	err_quit("Error: unable to open the filter database.");

    db_size = db.size();	// get database size
    printf("number of rules: %d\n", db_size);
    int block = 1024;
    int grid = 13;
    //int grid = db_size/block;
    //if(grid == 0)
	//grid = 1;
    // set to 0 for copying to device variable

    // prepare device variable
    hipMalloc((void**)&dev_filterSet, db_size * sizeof(filter));
    cudaCheckErrors("cudaMalloc1 fail");
    hipMalloc((void**)&dev_total, db_size * sizeof(dev_var));
    cudaCheckErrors("cudaMalloc2 fail");   
    hipMemset(dev_total, 0, db_size*sizeof(dev_total));

    cpy_dev = (dev_var*)malloc(db_size*sizeof(dev_var));
    // call kernel function
    g0conflict_start = cpuSecond();
    // copy to device
    hipMemcpy(dev_filterSet, db.filterSet_, db_size * sizeof(filter), hipMemcpyHostToDevice);
    cudaCheckErrors("cudaMemcpy1 fail");
    gpu_conflict_detect_0<<<grid, block>>>(dev_filterSet, db_size, dev_total);
    hipDeviceSynchronize();
    
    // copy device result ot host
    hipMemcpy(cpy_dev, dev_total, db_size * sizeof(dev_var), hipMemcpyDeviceToHost);
    int sum = 0;
    for(int tmp = 0; tmp < db_size; tmp++)
    {
        if(cpy_dev[tmp].set == 1)
	{
	    //printf("threadIdx:%d\t", tmp);
	    sum += cpy_dev[tmp].result;   
	}	
	//printf("srtIPlen:  %d\n", db.filterSet_[tmp].srcIPLen_);
        //printf("destIPlen: %d\n", db.filterSet_[tmp].destIPLen_);
        //system("read var1");
    }
    g0conflict_time = cpuSecond() - g0conflict_start;
    cout << "m0_detection_time: " << g0conflict_time << " usec" << endl;
    printf("m0 conflict detection: %d\n", sum);
    free(cpy_dev);
    hipFree(dev_filterSet);
    hipFree(dev_total);

    // sort filter with srcIPlen & desIPlen
    sort_start = cpuSecond();
    bubble_sort(db, db_size);
    sort_time = cpuSecond() - sort_start;
    cout << "sort_time: " << sort_time << " usec" << endl;

    // prepare device variable
    hipMalloc((void**)&dev_filterSet, db_size * sizeof(filter));
    cudaCheckErrors("cudaMalloc1 fail");
    hipMalloc((void**)&dev_total, db_size * sizeof(dev_var));
    cudaCheckErrors("cudaMalloc2 fail");   
    hipMemset(dev_total, 0, db_size*sizeof(dev_total));

    cpy_dev = (dev_var*)malloc(db_size*sizeof(dev_var));
    // call kernel function
    g1conflict_start = cpuSecond();
    // copy to device
    hipMemcpy(dev_filterSet, db.filterSet_, db_size * sizeof(filter), hipMemcpyHostToDevice);
    cudaCheckErrors("cudaMemcpy1 fail");
    gpu_conflict_detect_1<<<grid, block>>>(dev_filterSet, db_size, dev_total);
    hipDeviceSynchronize();
    
    // copy device result ot host
    hipMemcpy(cpy_dev, dev_total, db_size * sizeof(dev_var), hipMemcpyDeviceToHost);
    sum = 0;
    for(int tmp = 0; tmp < db_size; tmp++)
    {
        if(cpy_dev[tmp].set == 1)
	{
	    //printf("threadIdx:%d\t", tmp);
	    sum += cpy_dev[tmp].result;   
	}	
	//printf("srtIPlen:  %d\n", db.filterSet_[tmp].srcIPLen_);
        //printf("destIPlen: %d\n", db.filterSet_[tmp].destIPLen_);
        //system("read var1");
    }
    g1conflict_time = cpuSecond() - g1conflict_start;
    cout << "m1_detection_time: " << g1conflict_time << " usec" << endl;
    printf("m1 conflict detection: %d\n", sum);
    free(cpy_dev);
    hipFree(dev_filterSet);
    hipFree(dev_total);

    // prepare device variable
    hipMalloc((void**)&dev_filterSet, db_size * sizeof(filter));
    cudaCheckErrors("cudaMalloc1 fail");
    hipMalloc((void**)&dev_total, db_size * sizeof(dev_var));
    cudaCheckErrors("cudaMalloc2 fail");
    hipMemset(dev_total, 0, db_size*sizeof(dev_total));
 
    cpy_dev = (dev_var*)malloc(db_size*sizeof(dev_var));
      
    // call kernel function method 2
    g2conflict_start = cpuSecond();
    // copy to device
    hipMemcpy(dev_filterSet, db.filterSet_, db_size * sizeof(filter), hipMemcpyHostToDevice);
    cudaCheckErrors("cudaMemcpy1 fail");
    gpu_conflict_detect_2<<<grid, block>>>(dev_filterSet, db_size, dev_total);
    hipDeviceSynchronize();
    
    // copy device result ot host
    hipMemcpy(cpy_dev, dev_total, db_size * sizeof(dev_var), hipMemcpyDeviceToHost);
    sum = 0;
    for(int tmp = 0; tmp < db_size; tmp++)
    {
        if(cpy_dev[tmp].set == 1)
            sum += cpy_dev[tmp].result;
        //printf("srtIPlen:  %d\n", db.filterSet_[tmp].srcIPLen_);
        //printf("destIPlen: %d\n", db.filterSet_[tmp].destIPLen_);
        //system("read var1");
    }
    g2conflict_time = cpuSecond() - g2conflict_start;
    cout << "m2_detection_time: " << g2conflict_time << " usec" << endl;
    printf("m2 conflict detection: %d\n", sum);
    free(cpy_dev);
    hipFree(dev_filterSet);
    hipFree(dev_total);

// call host conflict function
    hconflict_start = cpuSecond();
    host_conflict_detect(db.filterSet_, db_size, total);
    hconflict_time = cpuSecond() - hconflict_start;
    cout << "host_detection_time: " << hconflict_time << " usec" << endl;
    printf("Host conflict detection: %d\n", total);
    
/*
fout.open("0714_512dbTCDA.txt", ios::app);   
fout<< hconflict_time<< "\t";
fout<< g1conflict_time<< "\t";
fout<< g2conflict_time<< "\n";
fout.close();
*/
    //free(cpy_dev);
    //hipFree(dev_filterSet);
    //hipFree(dev_total);
}


