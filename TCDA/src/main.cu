#include "hip/hip_runtime.h"
#include	<stdio.h>
#include	<string.h>
#include	"err_handler.h"
#include	"filter_db.h"
#include	"util.h"
#include    <sys/time.h>
#include    <fstream>
#include    <iostream>

#include    <hip/hip_runtime.h>

#define		BUF_SZ   256
using namespace std;
clsFilterDB   db;

#define cudaCheckErrors(msg) \
    do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                        msg, hipGetErrorString(__err), __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
    } while (0)

inline __host__ __device__ UINT64 genPrefix(UINT64 prefix, unsigned char len)
{
    UINT64 result;
    UINT mask_m;
    
    if (len == 0)
	return(0);
    else{
	// mask    = 0xFFFFFFFFFFFFFFFFLL << (64-len);
	mask_m    = 0xFFFFFFFF << (32-len);
        result  = prefix & mask_m;
        return(result);
    }
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec*1e6 + (double)tp.tv_usec);
}


void bubble_sort(clsFilterDB item, int len)
{
    int i, j;
    filter temp;
    for(i = 0; i < len - 1; i++)
        for(j = 0; j < len - i - 1; j++)
        {
            if(item.filterSet_[j].destIPLen_ > item.filterSet_[j+1].destIPLen_)
            {
                
                //printf("1.srcIP: %d\n", item.filterSet_[j].srcIPLen_);
                temp = item.filterSet_[j];
                item.filterSet_[j] = item.filterSet_[j+1];
                item.filterSet_[j+1] = temp;
                //printf("2.srcIP: %d\n", item.filterSet_[j].srcIPLen_);
            }
            if(item.filterSet_[j].srcIPLen_ > item.filterSet_[j+1].srcIPLen_)
            {
                //printf("1.srcIP: %d\n", item.filterSet_[j].srcIPLen_);
                temp = item.filterSet_[j];
                item.filterSet_[j] = item.filterSet_[j+1];
                item.filterSet_[j+1] = temp;
                //printf("2.srcIP: %d\n", item.filterSet_[j].srcIPLen_);
            }
            
        }
    
}

// distribute filters before sorting, to thread by order. (approach 1)
__global__ void gpu_conflict_detect_0(filter * const __restrict__ filterSet, int rule_size, int* d_total)
{
    //printf("blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x, threadIdx.x);
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int threadSize = gridDim.x * blockDim.x;
    int base = rule_size/threadSize+1;
    int start = threadID*base;
    int total = 0;
    __shared__ int shm[12*1024];
    shm[threadID] = 0;
    //devTotal[start].set = 1;
    //devTotal[threadID].set = 1;
    //devTotal[threadID].result = 0;
    //printf("threadSize:%d,%d\n", threadSize, rule_size/threadSize + 1);
    //for(int i=threadID; i<rule_size; i+=threadSize){  
    //if(threadID > rule_size) return;
    for (int i=start; i<(start+base) && i<rule_size ; i++) {
	//devTotal[threadID].set = 1;        
        for (int j= 0; j<rule_size; j++) {
	    //if(j>i) return;
            if( filterSet[i].srcIPLen_ > filterSet[j].srcIPLen_ ){
                if( genPrefix(filterSet[i].srcIP_, filterSet[j].srcIPLen_) == filterSet[j].srcIP_ ){
                    if(( (!filterSet[i].pro_num_ || !filterSet[j].pro_num_) || (filterSet[i].pro_num_ == filterSet[j].pro_num_ )) ) {
                        if (
                            (
                             
                             ((filterSet[i].destPortleft_ <= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ >= filterSet[j].destPortleft_))   ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ <= filterSet[j].destPortright_)) ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_))
                             
                             )
                            &&
                            (
                             ((filterSet[i].srcPortleft_ <= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ >= filterSet[j].srcPortleft_))   ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortleft_) && (filterSet[i].srcPortright_ <= filterSet[j].srcPortright_)) ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ <= filterSet[j].srcPortleft_))
                             
                             )
                            )
                        {
                            
                            
                            if (filterSet[i].destIPLen_ > filterSet[j].destIPLen_ ){
                                if ( genPrefix(filterSet[i].destIP_, filterSet[j].destIPLen_) == filterSet[j].destIP_){
                                    //counter1++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        //devTotal[threadID].result++;
					total++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ < filterSet[j].destIPLen_){
                                if ( genPrefix(filterSet[j].destIP_, filterSet[i].destIPLen_) == filterSet[i].destIP_){
                                    //counter2++;
                                    /*
                                     if(
                                     ( (rule.filterSet_[i].destPortleft_ <= rule.filterSet_[j].destPortleft_) && (rule.filterSet_[i].destPortright_ >= rule.filterSet_[j].destPortright_) ) &&
                                     ( (rule.filterSet_[i].srcPortleft_  <= rule.filterSet_[j].srcPortleft_)  && (rule.filterSet_[i].srcPortright_  >= rule.filterSet_[j].srcPortright_) ) &&
                                     ( (rule.filterSet_[i].pro_num_ == rule.filterSet_[j].pro_num_) || (rule.filterSet_[i].pro_num_ == 0) )
                                     ){}
                                     else
                                     */
                                    //devTotal++;
                                    //devTotal[threadID].result++;
				    total++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ == filterSet[j].destIPLen_){
                                if ( filterSet[i].destIP_ == filterSet[j].destIP_){
                                    //counter3++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        //devTotal[threadID].result++;
					total++;
                                }
                            }
                        }
                    }
                }
            }
        }
    }

    shm[threadID] = total;

    __syncthreads(); // all thread in the block will wait here
    if(threadIdx.x == 0)
    {
        for(int i = 1; i < blockDim.x; i++)
	{
	   shm[blockIdx.x*blockDim.x] += shm[i+blockIdx.x*blockDim.x];
        }
        //printf("%d\t", shm[blockIdx.x*blockDim.x]);
        *d_total += shm[blockIdx.x*blockDim.x];
    }
    
}

// base on new, distribute filters to thread by order. (approach 1)
__global__ void gpu_conflict_detect_1(filter * const __restrict__ filterSet, int rule_size, int* d_total)
{
    //printf("blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x, threadIdx.x);
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int threadSize = gridDim.x * blockDim.x;
    int base = rule_size/threadSize+1;
    int start = threadID*base;
    int total = 0;
    __shared__ int shm[12*1024];
    shm[threadID] = 0;
    //devTotal[start].set = 1;
    //devTotal[threadID].set = 1;
    //devTotal[threadID].result = 0;
    //printf("threadSize:%d,%d\n", threadSize, rule_size/threadSize + 1);
    //for(int i=threadID; i<rule_size; i+=threadSize){  
    //if(threadID > rule_size) return;
    for (int i=start; i<(start+base) && i<rule_size ; i++) {
	//devTotal[threadID].set = 1;        
        for (int j= 0; j<i; j++) {
	    //if(j>i) return;
            if( filterSet[i].srcIPLen_ > filterSet[j].srcIPLen_ ){
                if( genPrefix(filterSet[i].srcIP_, filterSet[j].srcIPLen_) == filterSet[j].srcIP_ ){
                    if(( (!filterSet[i].pro_num_ || !filterSet[j].pro_num_) || (filterSet[i].pro_num_ == filterSet[j].pro_num_ )) ) {
                        if (
                            (
                             
                             ((filterSet[i].destPortleft_ <= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ >= filterSet[j].destPortleft_))   ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ <= filterSet[j].destPortright_)) ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_))
                             
                             )
                            &&
                            (
                             ((filterSet[i].srcPortleft_ <= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ >= filterSet[j].srcPortleft_))   ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortleft_) && (filterSet[i].srcPortright_ <= filterSet[j].srcPortright_)) ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ <= filterSet[j].srcPortleft_))
                             
                             )
                            )
                        {
                            
                            
                            if (filterSet[i].destIPLen_ > filterSet[j].destIPLen_ ){
                                if ( genPrefix(filterSet[i].destIP_, filterSet[j].destIPLen_) == filterSet[j].destIP_){
                                    //counter1++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        //devTotal[threadID].result++;
					total++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ < filterSet[j].destIPLen_){
                                if ( genPrefix(filterSet[j].destIP_, filterSet[i].destIPLen_) == filterSet[i].destIP_){
                                    //counter2++;
                                    /*
                                     if(
                                     ( (rule.filterSet_[i].destPortleft_ <= rule.filterSet_[j].destPortleft_) && (rule.filterSet_[i].destPortright_ >= rule.filterSet_[j].destPortright_) ) &&
                                     ( (rule.filterSet_[i].srcPortleft_  <= rule.filterSet_[j].srcPortleft_)  && (rule.filterSet_[i].srcPortright_  >= rule.filterSet_[j].srcPortright_) ) &&
                                     ( (rule.filterSet_[i].pro_num_ == rule.filterSet_[j].pro_num_) || (rule.filterSet_[i].pro_num_ == 0) )
                                     ){}
                                     else
                                     */
                                    //devTotal++;
                                    //devTotal[threadID].result++;
				    total++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ == filterSet[j].destIPLen_){
                                if ( filterSet[i].destIP_ == filterSet[j].destIP_){
                                    //counter3++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        //devTotal[threadID].result++;
					total++;
                                }
                            }
                        }
                    }
                }
            }
        }
    }

    shm[threadID] = total;

    __syncthreads(); // all thread in the block will wait here
    if(threadIdx.x == 0)
    {
        for(int i = 1; i < blockDim.x; i++)
	{
	   shm[blockIdx.x*blockDim.x] += shm[i+blockIdx.x*blockDim.x];
        }
        //printf("%d\t", shm[blockIdx.x*blockDim.x]);
        *d_total += shm[blockIdx.x*blockDim.x];
    }
    
}

// base on new, distribute filters to thread by work. (approach 2)
__global__ void gpu_conflict_detect_2(filter * const __restrict__ filterSet, int rule_size, int* d_total)
{
    //printf("blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x, threadIdx.x);
    int start = threadIdx.x * gridDim.x + blockIdx.x;
    int threadSize = gridDim.x * blockDim.x;
    int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int total = 0;
    
    __shared__ int shm[12*1024]; // 48KBis the max;
    shm[threadIndex] = 0;

    //devTotal[threadID].set = 1;
    //printf("threadSize:%d,%d\n", threadSize, rule_size/threadSize + 1);
    
    for (int i=start; i<rule_size; i+=threadSize) {
    //for(int i=threadID; i<rule_size; i+=threadSize){  
        //devTotal[threadIndex].set = 1;
        for (int j= 0; j<i; j++) {
	    //if(j>i) break;
            if( filterSet[i].srcIPLen_ > filterSet[j].srcIPLen_ ){
                if( genPrefix(filterSet[i].srcIP_, filterSet[j].srcIPLen_) == filterSet[j].srcIP_ ){
                    if(( (!filterSet[i].pro_num_ || !filterSet[j].pro_num_) || (filterSet[i].pro_num_ == filterSet[j].pro_num_ )) ) {
                        if (
                            (
                             
                             ((filterSet[i].destPortleft_ <= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ >= filterSet[j].destPortleft_))   ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ <= filterSet[j].destPortright_)) ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_))
                             
                             )
                            &&
                            (
                             ((filterSet[i].srcPortleft_ <= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ >= filterSet[j].srcPortleft_))   ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortleft_) && (filterSet[i].srcPortright_ <= filterSet[j].srcPortright_)) ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ <= filterSet[j].srcPortleft_))
                             
                             )
                            )
                        {
                            
                            
                            if (filterSet[i].destIPLen_ > filterSet[j].destIPLen_ ){
                                if ( genPrefix(filterSet[i].destIP_, filterSet[j].destIPLen_) == filterSet[j].destIP_){
                                    //counter1++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal[threadIndex].result++;
                                        //shm[threadIndex]++;
					total++;
					
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ < filterSet[j].destIPLen_){
                                if ( genPrefix(filterSet[j].destIP_, filterSet[i].destIPLen_) == filterSet[i].destIP_){
                                    //counter2++;
                                    /*
                                     if(
                                     ( (rule.filterSet_[i].destPortleft_ <= rule.filterSet_[j].destPortleft_) && (rule.filterSet_[i].destPortright_ >= rule.filterSet_[j].destPortright_) ) &&
                                     ( (rule.filterSet_[i].srcPortleft_  <= rule.filterSet_[j].srcPortleft_)  && (rule.filterSet_[i].srcPortright_  >= rule.filterSet_[j].srcPortright_) ) &&
                                     ( (rule.filterSet_[i].pro_num_ == rule.filterSet_[j].pro_num_) || (rule.filterSet_[i].pro_num_ == 0) )
                                     ){}
                                     else
                                     */
                                    //devTotal[threadIndex].result++;
                                    //shm[threadIndex]++;
				    total++;
				    
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ == filterSet[j].destIPLen_){
                                if ( filterSet[i].destIP_ == filterSet[j].destIP_){
                                    //counter3++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal[threadIndex].result++;
                                        //shm[threadIndex]++;
					total++;
					
					
                                }
                            }
                        }
			
                    }
                }
            }
        }
    }
    //printf("%d\t", total);
    shm[threadIndex] = total;
    

    __syncthreads(); // all thread in the block will wait here
    if(threadIdx.x == 0)
    {
        for(int i = 1; i < blockDim.x; i++)
	{
	   shm[blockIdx.x*blockDim.x] += shm[i+blockIdx.x*blockDim.x];
        }
        //printf("%d\t", shm[blockIdx.x*blockDim.x]);
        *d_total += shm[blockIdx.x*blockDim.x];
    }
    //devTotal[threadID].result += total;

    
}

/*
__global__ void gpu_conflict_detect_2(const filter * __restrict__ filterSet, int rule_size, dev_var* devTotal)
{
    //printf("blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x, threadIdx.x);
    int threadID = threadIdx.x * gridDim.x + blockIdx.x;
    int threadSize = gridDim.x * blockDim.x;
    int total = 0;
    
    //devTotal[threadID].set = 1;
    //printf("threadSize:%d,%d\n", threadSize, rule_size/threadSize + 1);
    for (int i=threadID; i<rule_size; i+=threadSize) {
    //for(int i=threadID; i<rule_size; i+=threadSize){  
        
        //devTotal[i].result = 0;
        for (int j= 0; j<rule_size ; j++) {
            if( __ldg(&filterSet[i].srcIPLen_) > __ldg(&filterSet[j].srcIPLen_) ){
                if( genPrefix(__ldg(&filterSet[i].srcIP_), __ldg(&filterSet[j].srcIPLen_)) == __ldg(&filterSet[j].srcIP_) ){
                    if(( (!__ldg(&filterSet[i].pro_num_) || !__ldg(&filterSet[j].pro_num_)) || (__ldg(&filterSet[i].pro_num_) == __ldg(&filterSet[j].pro_num_) )) ) {
                        if (
                            (
                             
                             ((__ldg(&filterSet[i].destPortleft_) <= __ldg(&filterSet[j].destPortright_)) && (__ldg(&filterSet[i].destPortleft_) >= __ldg(&filterSet[j].destPortleft_)))   ||
                             ((__ldg(&filterSet[i].destPortright_) >= __ldg(&filterSet[j].destPortleft_)) && (__ldg(&filterSet[i].destPortright_) <= __ldg(&filterSet[j].destPortright_))) ||
                             ((__ldg(&filterSet[i].destPortright_) >= __ldg(&filterSet[j].destPortright_)) && (__ldg(&filterSet[i].destPortleft_) <= __ldg(&filterSet[j].destPortleft_)))
                             
                             )
                            &&
                            (
                             ((__ldg(&filterSet[i].srcPortleft_) <= __ldg(&filterSet[j].srcPortright_)) && (__ldg(&filterSet[i].srcPortleft_) >= __ldg(&filterSet[j].srcPortleft_)))   ||
                             ((__ldg(&filterSet[i].srcPortright_) >= __ldg(&filterSet[j].srcPortleft_)) && (__ldg(&filterSet[i].srcPortright_) <= __ldg(&filterSet[j].srcPortright_))) ||
                             ((__ldg(&filterSet[i].srcPortright_) >= __ldg(&filterSet[j].srcPortright_)) && (__ldg(&filterSet[i].srcPortleft_) <= __ldg(&filterSet[j].srcPortleft_)))
                             
                             )
                            )
                        {
                            
                            
                            if (__ldg(&filterSet[i].destIPLen_) > __ldg(&filterSet[j].destIPLen_) ){
                                if ( genPrefix(__ldg(&filterSet[i].destIP_), __ldg(&filterSet[j].destIPLen_)) == __ldg(&filterSet[j].destIP_)){
                                    //counter1++;
                                    
                                    if(
                                       ( (__ldg(&filterSet[i].destPortleft_) <= __ldg(&filterSet[j].destPortleft_)) && (__ldg(&filterSet[i].destPortright_) >= __ldg(&filterSet[j].destPortright_)) ) &&
                                       ( (__ldg(&filterSet[i].srcPortleft_)  <= __ldg(&filterSet[j].srcPortleft_))  && (__ldg(&filterSet[i].srcPortright_)  >= __ldg(&filterSet[j].srcPortright_)) ) &&
                                       ( (__ldg(&filterSet[i].pro_num_) == __ldg(&filterSet[j].pro_num_)) || (__ldg(&filterSet[i].pro_num_) == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        //devTotal[threadID].result++;
					total++;
					
                                }
                            }
                            
                            if (__ldg(&filterSet[i].destIPLen_) < __ldg(&filterSet[j].destIPLen_)){
                                if ( genPrefix(__ldg(&filterSet[j].destIP_), __ldg(&filterSet[i].destIPLen_)) == __ldg(&filterSet[i].destIP_)){
                                    //counter2++;
                                    
                                     //if(
                                     //( (rule.__ldg(&filterSet_[i].destPortleft_) <= rule.__ldg(&filterSet_[j].destPortleft_)) && (rule.__ldg(&filterSet_[i].destPortright_) >= rule.__ldg(&filterSet_[j].destPortright_)) ) &&
                                     //( (rule.__ldg(&filterSet_[i].srcPortleft_)  <= rule.__ldg(&filterSet_[j].srcPortleft_))  && (rule.__ldg(&filterSet_[i].srcPortright_)  >= rule.__ldg(&filterSet_[j].srcPortright_)) ) &&
                                     //( (rule.__ldg(&filterSet_[i].pro_num_) == rule.__ldg(&filterSet_[j].pro_num_)) || (rule.__ldg(&filterSet_[i].pro_num_) == 0) )
                                     //){}
                                     //else
                                     
                                    //devTotal++;
                                    //devTotal[threadID].result++;
				    total++;
				    
                                }
                            }
                            
                            if (__ldg(&filterSet[i].destIPLen_) == __ldg(&filterSet[j].destIPLen_)){
                                if ( __ldg(&filterSet[i].destIP_) == __ldg(&filterSet[j].destIP_)){
                                    //counter3++;
                                    
                                    if(
                                       ( (__ldg(&filterSet[i].destPortleft_) <= __ldg(&filterSet[j].destPortleft_)) && (__ldg(&filterSet[i].destPortright_) >= __ldg(&filterSet[j].destPortright_)) ) &&
                                       ( (__ldg(&filterSet[i].srcPortleft_)  <= __ldg(&filterSet[j].srcPortleft_))  && (__ldg(&filterSet[i].srcPortright_)  >= __ldg(&filterSet[j].srcPortright_)) ) &&
                                       ( (__ldg(&filterSet[i].pro_num_) == __ldg(&filterSet[j].pro_num_)) || (__ldg(&filterSet[i].pro_num_) == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        //devTotal[threadID].result++;
					total++;
					
					
                                }
                            }
                        }
			
                    }
                }
            }
        }
    }
    //devTotal[threadID].set = 1;
    //devTotal[threadID].result += total;
    //__syncthreads();
    //devTotal[threadIdx.x].set = 1;
    //devTotal[threadIdx.x].result += shm[threadIdx.x];
    //printf("%d\t", devTotal[threadIdx.x].result);
    
}
*/

void host_conflict_detect(filter *filterSet, int rule_size, int &total)
{
    for (int i=0; i<rule_size; i++)	{
        for (int j= 0; j<i; j++) {
            if( (filterSet[i].srcIPLen_ > filterSet[j].srcIPLen_ ) && ( genPrefix(filterSet[i].srcIP_, filterSet[j].srcIPLen_) == filterSet[j].srcIP_ ) ){
                if(( (!filterSet[i].pro_num_ || !filterSet[j].pro_num_) || (filterSet[i].pro_num_ == filterSet[j].pro_num_ )) ) {
                    if (
                        (
                         
                         ((filterSet[i].destPortleft_ <= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ >= filterSet[j].destPortleft_))   ||
                         ((filterSet[i].destPortright_ >= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ <= filterSet[j].destPortright_)) ||
                         ((filterSet[i].destPortright_ >= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_))
                         
                        )
                        &&
                        (
                         ((filterSet[i].srcPortleft_ <= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ >= filterSet[j].srcPortleft_))   ||
                         ((filterSet[i].srcPortright_ >= filterSet[j].srcPortleft_) && (filterSet[i].srcPortright_ <= filterSet[j].srcPortright_)) ||
                         ((filterSet[i].srcPortright_ >= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ <= filterSet[j].srcPortleft_))
                         
                         )
                        )
                    {
                        
                        
                        if (filterSet[i].destIPLen_ > filterSet[j].destIPLen_ ){
                            if ( genPrefix(filterSet[i].destIP_, filterSet[j].destIPLen_) == filterSet[j].destIP_){
                                //counter1++;
                                
                                if(
                                   ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                   ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                   ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                   ){}
                                else
                                    total++;
                            }
                        }
                        
                        if (filterSet[i].destIPLen_ < filterSet[j].destIPLen_){
                            if ( genPrefix(filterSet[j].destIP_, filterSet[i].destIPLen_) == filterSet[i].destIP_){
                                //counter2++;
                                /*
                                 if(
                                 ( (rule.filterSet_[i].destPortleft_ <= rule.filterSet_[j].destPortleft_) && (rule.filterSet_[i].destPortright_ >= rule.filterSet_[j].destPortright_) ) &&
                                 ( (rule.filterSet_[i].srcPortleft_  <= rule.filterSet_[j].srcPortleft_)  && (rule.filterSet_[i].srcPortright_  >= rule.filterSet_[j].srcPortright_) ) &&
                                 ( (rule.filterSet_[i].pro_num_ == rule.filterSet_[j].pro_num_) || (rule.filterSet_[i].pro_num_ == 0) )
                                 ){}
                                 else
                                 */
                                total++;
                            }
                        }
                        
                        if (filterSet[i].destIPLen_ == filterSet[j].destIPLen_){
                            if ( filterSet[i].destIP_ == filterSet[j].destIP_){
                                //counter3++;
                                
                                if(
                                   ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                   ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                   ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                   ){}
                                else
                                    total++;
                            }
                        }
                        
                    }
                }
            }
        }
    }
    
}


int main(int argc, char *argv[])
{

    ofstream fout;
    double sort_start, hconflict_start, g1conflict_start, g2conflict_start, g0conflict_start;
    double sort_time, hconflict_time, g1conflict_time, g2conflict_time, g0conflict_time;
    int db_size;
    int total = 0;// for host computaion result
    int *d_total;// store device result
    int *h_total;// store host result

    // CUDA variable
    filter* dev_filterSet;	// store filter in device
    printf("%d byte\n", sizeof(filter));
    //insert filters
    if (db.loadFilters(argv[1]) == -1)
	err_quit("Error: unable to open the filter database.");

    db_size = db.size();	// get database size
    printf("number of rules: %d\n", db_size);
    int block = 512;
    int grid = 4;
    //int grid =( db_size + block - 1)/block;
    //if(grid == 0)
	//grid = 1;
    // set to 0 for copying to device variable

    //hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    //hipSetDevice(0);
    // prepare device variable

/*
// ################ Prepare call M0 funtion ###################
    hipMalloc((void**)&dev_filterSet, db_size * sizeof(filter));
    cudaCheckErrors("cudaMalloc1 fail");
    hipMalloc((void**)&d_total, sizeof(int));
    cudaCheckErrors("cudaMalloc2 fail");   
    hipMemset(d_total, 0, sizeof(int));

    h_total = (int*)malloc(sizeof(int));
    // call kernel function
    g0conflict_start = cpuSecond();
    // copy to device
    hipMemcpy(dev_filterSet, db.filterSet_, db_size * sizeof(filter), hipMemcpyHostToDevice);
    cudaCheckErrors("cudaMemcpy1 fail");
    gpu_conflict_detect_0<<<grid, block>>>(dev_filterSet, db_size, d_total);
    hipDeviceSynchronize();
    
    // copy device result ot host
    hipMemcpy(h_total, d_total, sizeof(int), hipMemcpyDeviceToHost);

    g0conflict_time = cpuSecond() - g0conflict_start;
    cout << "m0_detection_time: " << g0conflict_time << " usec" << endl;
    printf("m0 conflict detection: %d\n", *h_total);
    free(h_total);
    hipFree(dev_filterSet);
    hipFree(d_total);
*/


    // sort filter with srcIPlen & desIPlen
    sort_start = cpuSecond();
    bubble_sort(db, db_size);
    sort_time = cpuSecond() - sort_start;
    //cout << "sort_time: " << sort_time << " usec" << endl;

/*
// ################ Prepare call M1 funtion ###################
    // prepare device variable
    hipMalloc((void**)&dev_filterSet, db_size * sizeof(filter));
    cudaCheckErrors("cudaMalloc1 fail");
    hipMalloc((void**)&d_total, sizeof(int));
    cudaCheckErrors("cudaMalloc2 fail");
    hipMemset(d_total, 0, sizeof(int));
    h_total = (int*)malloc(sizeof(int));
    // call kernel function
    g1conflict_start = cpuSecond();
    // copy to device
    hipMemcpy(dev_filterSet, db.filterSet_, db_size * sizeof(filter), hipMemcpyHostToDevice);
    cudaCheckErrors("cudaMemcpy1 fail");
    gpu_conflict_detect_1<<<grid, block>>>(dev_filterSet, db_size, d_total);
    hipDeviceSynchronize();
    
    // copy device result ot host
    hipMemcpy(h_total, d_total, sizeof(int), hipMemcpyDeviceToHost);
   
    g1conflict_time = cpuSecond() - g1conflict_start;
    cout << "m1_detection_time: " << g1conflict_time << " usec" << endl;
    printf("m1 conflict detection: %d\n", *h_total);
    free(h_total);
    hipFree(dev_filterSet);
    hipFree(d_total);
*/
// ################ Prepare call M2 funtion ###################
    // prepare device variable
    hipMalloc((void**)&dev_filterSet, db_size * sizeof(filter));
    cudaCheckErrors("cudaMalloc1 fail");
    hipMalloc((void**)&d_total, sizeof(int));
    cudaCheckErrors("cudaMalloc2 fail");
    hipMemset(d_total, 0, sizeof(int));
    h_total = (int*)malloc(sizeof(int));
      
    // call kernel function method 2
    g2conflict_start = cpuSecond();
    // copy to device
    hipMemcpy(dev_filterSet, db.filterSet_, db_size * sizeof(filter), hipMemcpyHostToDevice);
    cudaCheckErrors("cudaMemcpy1 fail");
    gpu_conflict_detect_2<<<grid, block>>>(dev_filterSet, db_size, d_total);
    hipDeviceSynchronize();
    
    // copy device result ot host
    hipMemcpy(h_total, d_total, sizeof(int), hipMemcpyDeviceToHost);
    
    g2conflict_time = cpuSecond() - g2conflict_start;
    cout << "m2_detection_time: " << g2conflict_time << " usec" << endl;
    printf("m2 conflict detection: %d\n", *h_total);
    free(h_total);
    hipFree(dev_filterSet);
    hipFree(d_total);


    hipDeviceReset();

// ################ Prepare call host funtion ###################
    hconflict_start = cpuSecond();
    host_conflict_detect(db.filterSet_, db_size, total);
    hconflict_time = cpuSecond() - hconflict_start;
    cout.setf(ios::fixed);
    cout << std::noshowpoint << "host_detection_time: " << hconflict_time << " usec" << endl;
    printf("Host conflict detection: %d\n", total);
    
/*
fout.open("0714_512dbTCDA.txt", ios::app);   
fout<< hconflict_time<< "\t";
fout<< g1conflict_time<< "\t";
fout<< g2conflict_time<< "\n";
fout.close();
*/
    //free(cpy_dev);
    //hipFree(dev_filterSet);
    //hipFree(dev_total);
}


