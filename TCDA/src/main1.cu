#include "hip/hip_runtime.h"
#include	<stdio.h>
#include	<string.h>
#include	"err_handler.h"
#include	"filter_db.h"
#include	"util.h"
#include    <sys/time.h>
#include    <fstream>
#include    <iostream>

#include    <hip/hip_runtime.h>

#define		BUF_SZ   256
using namespace std;
clsFilterDB   db;

#define cudaCheckErrors(msg) \
    do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                        msg, hipGetErrorString(__err), __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
    } while (0)

__inline__ __device__ int warpReduceSum(int val)
{
    for(int offset = warpSize/2; offset > 0; offset /= 2)
    {
	val += __shfl_down(val, offset);
        //printf("%d\t", val);
    }
    return val;
}

__inline__ __device__ int blockReduceSum(int val)
{
    static __shared__ int shared[32];
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize; 
    
    val = warpReduceSum(val); // warp partial sum
    
    if(lane == 0) shared[wid] = val; // store each warp partial sum
    
    __syncthreads();

    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;
    if(wid==0) val = warpReduceSum(val);
    
    return val;
}

__inline__ __host__ __device__ UINT64 genPrefix(UINT64 prefix, unsigned char len)
{
    register UINT64 result;
    register UINT mask_m;
    
    if (len == 0)
	return(0);
    else{
	// mask    = 0xFFFFFFFFFFFFFFFFLL << (64-len);
	mask_m    = 0xFFFFFFFF << (32-len);
        result  = prefix & mask_m;
        return(result);
    }
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec*1e6 + (double)tp.tv_usec);
}


void bubble_sort(clsFilterDB item, int len)
{
    int i, j;
    filter temp;
    for(i = 0; i < len - 1; i++)
        for(j = 0; j < len - i - 1; j++)
        {
            if(item.filterSet_[j].destIPLen_ > item.filterSet_[j+1].destIPLen_)
            {
                
                //printf("1.srcIP: %d\n", item.filterSet_[j].srcIPLen_);
                temp = item.filterSet_[j];
                item.filterSet_[j] = item.filterSet_[j+1];
                item.filterSet_[j+1] = temp;
                //printf("2.srcIP: %d\n", item.filterSet_[j].srcIPLen_);
            }
            if(item.filterSet_[j].srcIPLen_ > item.filterSet_[j+1].srcIPLen_)
            {
                //printf("1.srcIP: %d\n", item.filterSet_[j].srcIPLen_);
                temp = item.filterSet_[j];
                item.filterSet_[j] = item.filterSet_[j+1];
                item.filterSet_[j+1] = temp;
                //printf("2.srcIP: %d\n", item.filterSet_[j].srcIPLen_);
            }
            
        }
    
}

// distribute filters before sorting, to thread by order. (approach 0)
__global__ void gpu_conflict_detect_0(filter * const __restrict__ filterSet, int rule_size, int* d_total)
{
    //printf("blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x, threadIdx.x);
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int threadSize = gridDim.x * blockDim.x;
    int base = rule_size/threadSize+1;
    int start = threadID*base;
    int total = 0;

    //printf("threadSize:%d,%d\n", threadSize, rule_size/threadSize + 1);
    //for(int i=threadID; i<rule_size; i+=threadSize){  
    if(threadID > rule_size) return;
    for (int i=start; i<(start+base) && i<rule_size ; i++) {
	//devTotal[threadID].set = 1;    
        for (int j= 0; j<rule_size; j++) {
	    //if(j>=i) return;
            if( filterSet[i].srcIPLen_ > filterSet[j].srcIPLen_ ){
                if( genPrefix(filterSet[i].srcIP_, filterSet[j].srcIPLen_) == filterSet[j].srcIP_ ){
                    if(( (!filterSet[i].pro_num_ || !filterSet[j].pro_num_) || (filterSet[i].pro_num_ == filterSet[j].pro_num_ )) ) {
                        if (
                            (
                             
                             ((filterSet[i].destPortleft_ <= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ >= filterSet[j].destPortleft_))   ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ <= filterSet[j].destPortright_)) ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_))
                             
                             )
                            &&
                            (
                             ((filterSet[i].srcPortleft_ <= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ >= filterSet[j].srcPortleft_))   ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortleft_) && (filterSet[i].srcPortright_ <= filterSet[j].srcPortright_)) ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ <= filterSet[j].srcPortleft_))
                             
                             )
                            )
                        {
                            
                            
                            if (filterSet[i].destIPLen_ > filterSet[j].destIPLen_ ){
                                if ( genPrefix(filterSet[i].destIP_, filterSet[j].destIPLen_) == filterSet[j].destIP_){
                                    //counter1++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        //devTotal[threadID].result++;
					total++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ < filterSet[j].destIPLen_){
                                if ( genPrefix(filterSet[j].destIP_, filterSet[i].destIPLen_) == filterSet[i].destIP_){
                                    //counter2++;
                                    /*
                                     if(
                                     ( (rule.filterSet_[i].destPortleft_ <= rule.filterSet_[j].destPortleft_) && (rule.filterSet_[i].destPortright_ >= rule.filterSet_[j].destPortright_) ) &&
                                     ( (rule.filterSet_[i].srcPortleft_  <= rule.filterSet_[j].srcPortleft_)  && (rule.filterSet_[i].srcPortright_  >= rule.filterSet_[j].srcPortright_) ) &&
                                     ( (rule.filterSet_[i].pro_num_ == rule.filterSet_[j].pro_num_) || (rule.filterSet_[i].pro_num_ == 0) )
                                     ){}
                                     else
                                    */
                                    //devTotal++;
                                    //devTotal[threadID].result++;
				    total++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ == filterSet[j].destIPLen_){
                                if ( filterSet[i].destIP_ == filterSet[j].destIP_){
                                    //counter3++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        //devTotal[threadID].result++;
					total++;
                                }
                            }
                        }
                    }
                }
            }
        }

    }
    total = blockReduceSum(total);

    if(threadIdx.x == 0)
	d_total[blockIdx.x] = total;

}

// base on new, distribute filters to thread by order. (approach 1)
__global__ void gpu_conflict_detect_1(filter * const __restrict__ filterSet, int rule_size, int* d_total)
{
    //printf("blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x, threadIdx.x);
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int threadSize = gridDim.x * blockDim.x;
    int base = rule_size/threadSize+1;
    int start = threadID*base;
    int total = 0;
   
    //for(int i=threadID; i<rule_size; i+=threadSize){  
    if(threadID > rule_size) return;
    for (int i=start; i<(start+base) && i<rule_size ; i++) {
	//devTotal[threadID].set = 1;      
        for (int j= 0; j<i; j++) {
	    //if(j>=i) break;
            if( filterSet[i].srcIPLen_ > filterSet[j].srcIPLen_ ){
                if( genPrefix(filterSet[i].srcIP_, filterSet[j].srcIPLen_) == filterSet[j].srcIP_ ){
                    if(( (!filterSet[i].pro_num_ || !filterSet[j].pro_num_) || (filterSet[i].pro_num_ == filterSet[j].pro_num_ )) ) {
                        if (
                            (
                             
                             ((filterSet[i].destPortleft_ <= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ >= filterSet[j].destPortleft_))   ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ <= filterSet[j].destPortright_)) ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_))
                             
                             )
                            &&
                            (
                             ((filterSet[i].srcPortleft_ <= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ >= filterSet[j].srcPortleft_))   ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortleft_) && (filterSet[i].srcPortright_ <= filterSet[j].srcPortright_)) ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ <= filterSet[j].srcPortleft_))
                             
                             )
                            )
                        {
                            
                            
                            if (filterSet[i].destIPLen_ > filterSet[j].destIPLen_ ){
                                if ( genPrefix(filterSet[i].destIP_, filterSet[j].destIPLen_) == filterSet[j].destIP_){
                                    //counter1++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        //devTotal[threadID].result++;
					total++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ < filterSet[j].destIPLen_){
                                if ( genPrefix(filterSet[j].destIP_, filterSet[i].destIPLen_) == filterSet[i].destIP_){
                                    //counter2++;
                                    /*
                                     if(
                                     ( (rule.filterSet_[i].destPortleft_ <= rule.filterSet_[j].destPortleft_) && (rule.filterSet_[i].destPortright_ >= rule.filterSet_[j].destPortright_) ) &&
                                     ( (rule.filterSet_[i].srcPortleft_  <= rule.filterSet_[j].srcPortleft_)  && (rule.filterSet_[i].srcPortright_  >= rule.filterSet_[j].srcPortright_) ) &&
                                     ( (rule.filterSet_[i].pro_num_ == rule.filterSet_[j].pro_num_) || (rule.filterSet_[i].pro_num_ == 0) )
                                     ){}
                                     else
                                     */
                                    //devTotal++;
                                    //devTotal[threadID].result++;
				    total++;
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ == filterSet[j].destIPLen_){
                                if ( filterSet[i].destIP_ == filterSet[j].destIP_){
                                    //counter3++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        //devTotal[threadID].result++;
					total++;
                                }
                            }
                        }
                    }
                }
            }
        }
    }
    total = blockReduceSum(total);
    if(threadIdx.x == 0)
	d_total[blockIdx.x] = total;
}

// base on new, distribute filters to thread by work. (approach 2)
__global__ void gpu_conflict_detect_2(filter * const __restrict__ filterSet, int rule_size, int* d_total)
{
    //printf("blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x, threadIdx.x);
    int start = threadIdx.x * gridDim.x + blockIdx.x;
    int threadSize = gridDim.x * blockDim.x;
    int total = 0;
    int i, j;
    //printf("threadSize:%d,%d\n", threadSize, rule_size/threadSize + 1);
    if(start > rule_size) return;
    for (i=start; i<rule_size; i+=threadSize) {
    //for(int i=threadID; i<rule_size; i+=threadSize){  
        //devTotal[threadIndex].set = 1;
        for (j= 0; j<i; j++) {
	    //if(j>=i) break;
            if( filterSet[i].srcIPLen_ > filterSet[j].srcIPLen_ ){
                if( genPrefix(filterSet[i].srcIP_, filterSet[j].srcIPLen_) == filterSet[j].srcIP_ ){
                    if(( (!filterSet[i].pro_num_ || !filterSet[j].pro_num_) || (filterSet[i].pro_num_ == filterSet[j].pro_num_ )) ) {
                        if (
                            (
                             
                             ((filterSet[i].destPortleft_ <= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ >= filterSet[j].destPortleft_))   ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ <= filterSet[j].destPortright_)) ||
                             ((filterSet[i].destPortright_ >= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_))
                             
                             )
                            &&
                            (
                             ((filterSet[i].srcPortleft_ <= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ >= filterSet[j].srcPortleft_))   ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortleft_) && (filterSet[i].srcPortright_ <= filterSet[j].srcPortright_)) ||
                             ((filterSet[i].srcPortright_ >= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ <= filterSet[j].srcPortleft_))
                             
                             )
                            )
                        {
                            
                            
                            if (filterSet[i].destIPLen_ > filterSet[j].destIPLen_ ){
                                if ( genPrefix(filterSet[i].destIP_, filterSet[j].destIPLen_) == filterSet[j].destIP_){
                                    //counter1++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal[threadIndex].result++;
                                        //shm[threadIndex]++;
					total++;
					
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ < filterSet[j].destIPLen_){
                                if ( genPrefix(filterSet[j].destIP_, filterSet[i].destIPLen_) == filterSet[i].destIP_){
                                    //counter2++;
                                    /*
                                     if(
                                     ( (rule.filterSet_[i].destPortleft_ <= rule.filterSet_[j].destPortleft_) && (rule.filterSet_[i].destPortright_ >= rule.filterSet_[j].destPortright_) ) &&
                                     ( (rule.filterSet_[i].srcPortleft_  <= rule.filterSet_[j].srcPortleft_)  && (rule.filterSet_[i].srcPortright_  >= rule.filterSet_[j].srcPortright_) ) &&
                                     ( (rule.filterSet_[i].pro_num_ == rule.filterSet_[j].pro_num_) || (rule.filterSet_[i].pro_num_ == 0) )
                                     ){}
                                     else
                                     */
                                    //devTotal[threadIndex].result++;
                                    //shm[threadIndex]++;
				    total++;
				    
                                }
                            }
                            
                            if (filterSet[i].destIPLen_ == filterSet[j].destIPLen_){
                                if ( filterSet[i].destIP_ == filterSet[j].destIP_){
                                    //counter3++;
                                    
                                    if(
                                       ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                       ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                       ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                       ){}
                                    else
                                        //devTotal[threadIndex].result++;
                                        //shm[threadIndex]++;
					total++;
					
					
                                }
                            }
                        }
			
                    }
                }
            }
        }
    }
    //printf("%d\t", total);
    total = blockReduceSum(total);
    
    if(threadIdx.x == 0)
	d_total[blockIdx.x] = total;

}

/*
__global__ void gpu_conflict_detect_2(const filter * __restrict__ filterSet, int rule_size, dev_var* devTotal)
{
    //printf("blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x, threadIdx.x);
    int threadID = threadIdx.x * gridDim.x + blockIdx.x;
    int threadSize = gridDim.x * blockDim.x;
    int total = 0;
    
    //devTotal[threadID].set = 1;
    //printf("threadSize:%d,%d\n", threadSize, rule_size/threadSize + 1);
    for (int i=threadID; i<rule_size; i+=threadSize) {
    //for(int i=threadID; i<rule_size; i+=threadSize){  
        
        //devTotal[i].result = 0;
        for (int j= 0; j<rule_size ; j++) {
            if( __ldg(&filterSet[i].srcIPLen_) > __ldg(&filterSet[j].srcIPLen_) ){
                if( genPrefix(__ldg(&filterSet[i].srcIP_), __ldg(&filterSet[j].srcIPLen_)) == __ldg(&filterSet[j].srcIP_) ){
                    if(( (!__ldg(&filterSet[i].pro_num_) || !__ldg(&filterSet[j].pro_num_)) || (__ldg(&filterSet[i].pro_num_) == __ldg(&filterSet[j].pro_num_) )) ) {
                        if (
                            (
                             
                             ((__ldg(&filterSet[i].destPortleft_) <= __ldg(&filterSet[j].destPortright_)) && (__ldg(&filterSet[i].destPortleft_) >= __ldg(&filterSet[j].destPortleft_)))   ||
                             ((__ldg(&filterSet[i].destPortright_) >= __ldg(&filterSet[j].destPortleft_)) && (__ldg(&filterSet[i].destPortright_) <= __ldg(&filterSet[j].destPortright_))) ||
                             ((__ldg(&filterSet[i].destPortright_) >= __ldg(&filterSet[j].destPortright_)) && (__ldg(&filterSet[i].destPortleft_) <= __ldg(&filterSet[j].destPortleft_)))
                             
                             )
                            &&
                            (
                             ((__ldg(&filterSet[i].srcPortleft_) <= __ldg(&filterSet[j].srcPortright_)) && (__ldg(&filterSet[i].srcPortleft_) >= __ldg(&filterSet[j].srcPortleft_)))   ||
                             ((__ldg(&filterSet[i].srcPortright_) >= __ldg(&filterSet[j].srcPortleft_)) && (__ldg(&filterSet[i].srcPortright_) <= __ldg(&filterSet[j].srcPortright_))) ||
                             ((__ldg(&filterSet[i].srcPortright_) >= __ldg(&filterSet[j].srcPortright_)) && (__ldg(&filterSet[i].srcPortleft_) <= __ldg(&filterSet[j].srcPortleft_)))
                             
                             )
                            )
                        {
                            
                            
                            if (__ldg(&filterSet[i].destIPLen_) > __ldg(&filterSet[j].destIPLen_) ){
                                if ( genPrefix(__ldg(&filterSet[i].destIP_), __ldg(&filterSet[j].destIPLen_)) == __ldg(&filterSet[j].destIP_)){
                                    //counter1++;
                                    
                                    if(
                                       ( (__ldg(&filterSet[i].destPortleft_) <= __ldg(&filterSet[j].destPortleft_)) && (__ldg(&filterSet[i].destPortright_) >= __ldg(&filterSet[j].destPortright_)) ) &&
                                       ( (__ldg(&filterSet[i].srcPortleft_)  <= __ldg(&filterSet[j].srcPortleft_))  && (__ldg(&filterSet[i].srcPortright_)  >= __ldg(&filterSet[j].srcPortright_)) ) &&
                                       ( (__ldg(&filterSet[i].pro_num_) == __ldg(&filterSet[j].pro_num_)) || (__ldg(&filterSet[i].pro_num_) == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        //devTotal[threadID].result++;
					total++;
					
                                }
                            }
                            
                            if (__ldg(&filterSet[i].destIPLen_) < __ldg(&filterSet[j].destIPLen_)){
                                if ( genPrefix(__ldg(&filterSet[j].destIP_), __ldg(&filterSet[i].destIPLen_)) == __ldg(&filterSet[i].destIP_)){
                                    //counter2++;
                                    
                                     //if(
                                     //( (rule.__ldg(&filterSet_[i].destPortleft_) <= rule.__ldg(&filterSet_[j].destPortleft_)) && (rule.__ldg(&filterSet_[i].destPortright_) >= rule.__ldg(&filterSet_[j].destPortright_)) ) &&
                                     //( (rule.__ldg(&filterSet_[i].srcPortleft_)  <= rule.__ldg(&filterSet_[j].srcPortleft_))  && (rule.__ldg(&filterSet_[i].srcPortright_)  >= rule.__ldg(&filterSet_[j].srcPortright_)) ) &&
                                     //( (rule.__ldg(&filterSet_[i].pro_num_) == rule.__ldg(&filterSet_[j].pro_num_)) || (rule.__ldg(&filterSet_[i].pro_num_) == 0) )
                                     //){}
                                     //else
                                     
                                    //devTotal++;
                                    //devTotal[threadID].result++;
				    total++;
				    
                                }
                            }
                            
                            if (__ldg(&filterSet[i].destIPLen_) == __ldg(&filterSet[j].destIPLen_)){
                                if ( __ldg(&filterSet[i].destIP_) == __ldg(&filterSet[j].destIP_)){
                                    //counter3++;
                                    
                                    if(
                                       ( (__ldg(&filterSet[i].destPortleft_) <= __ldg(&filterSet[j].destPortleft_)) && (__ldg(&filterSet[i].destPortright_) >= __ldg(&filterSet[j].destPortright_)) ) &&
                                       ( (__ldg(&filterSet[i].srcPortleft_)  <= __ldg(&filterSet[j].srcPortleft_))  && (__ldg(&filterSet[i].srcPortright_)  >= __ldg(&filterSet[j].srcPortright_)) ) &&
                                       ( (__ldg(&filterSet[i].pro_num_) == __ldg(&filterSet[j].pro_num_)) || (__ldg(&filterSet[i].pro_num_) == 0) )
                                       ){}
                                    else
                                        //devTotal++;
                                        //devTotal[threadID].result++;
					total++;
					
					
                                }
                            }
                        }
			
                    }
                }
            }
        }
    }
    //devTotal[threadID].set = 1;
    //devTotal[threadID].result += total;
    //__syncthreads();
    //devTotal[threadIdx.x].set = 1;
    //devTotal[threadIdx.x].result += shm[threadIdx.x];
    //printf("%d\t", devTotal[threadIdx.x].result);
    
}
*/

void host_conflict_detect(filter *filterSet, int rule_size, int &total)
{
    for (int i=0; i<rule_size; i++)	{
        for (int j= 0; j<i; j++) {
            if( (filterSet[i].srcIPLen_ > filterSet[j].srcIPLen_ ) && ( genPrefix(filterSet[i].srcIP_, filterSet[j].srcIPLen_) == filterSet[j].srcIP_ ) ){
                if(( (!filterSet[i].pro_num_ || !filterSet[j].pro_num_) || (filterSet[i].pro_num_ == filterSet[j].pro_num_ )) ) {
                    if (
                        (
                         
                         ((filterSet[i].destPortleft_ <= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ >= filterSet[j].destPortleft_))   ||
                         ((filterSet[i].destPortright_ >= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ <= filterSet[j].destPortright_)) ||
                         ((filterSet[i].destPortright_ >= filterSet[j].destPortright_) && (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_))
                         
                        )
                        &&
                        (
                         ((filterSet[i].srcPortleft_ <= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ >= filterSet[j].srcPortleft_))   ||
                         ((filterSet[i].srcPortright_ >= filterSet[j].srcPortleft_) && (filterSet[i].srcPortright_ <= filterSet[j].srcPortright_)) ||
                         ((filterSet[i].srcPortright_ >= filterSet[j].srcPortright_) && (filterSet[i].srcPortleft_ <= filterSet[j].srcPortleft_))
                         
                         )
                        )
                    {
                        
                        
                        if (filterSet[i].destIPLen_ > filterSet[j].destIPLen_ ){
                            if ( genPrefix(filterSet[i].destIP_, filterSet[j].destIPLen_) == filterSet[j].destIP_){
                                //counter1++;
                                
                                if(
                                   ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                   ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                   ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                   ){}
                                else
                                    total++;
                            }
                        }
                        
                        if (filterSet[i].destIPLen_ < filterSet[j].destIPLen_){
                            if ( genPrefix(filterSet[j].destIP_, filterSet[i].destIPLen_) == filterSet[i].destIP_){
                                //counter2++;
                                /*
                                 if(
                                 ( (rule.filterSet_[i].destPortleft_ <= rule.filterSet_[j].destPortleft_) && (rule.filterSet_[i].destPortright_ >= rule.filterSet_[j].destPortright_) ) &&
                                 ( (rule.filterSet_[i].srcPortleft_  <= rule.filterSet_[j].srcPortleft_)  && (rule.filterSet_[i].srcPortright_  >= rule.filterSet_[j].srcPortright_) ) &&
                                 ( (rule.filterSet_[i].pro_num_ == rule.filterSet_[j].pro_num_) || (rule.filterSet_[i].pro_num_ == 0) )
                                 ){}
                                 else
                                 */
                                total++;
                            }
                        }
                        
                        if (filterSet[i].destIPLen_ == filterSet[j].destIPLen_){
                            if ( filterSet[i].destIP_ == filterSet[j].destIP_){
                                //counter3++;
                                
                                if(
                                   ( (filterSet[i].destPortleft_ <= filterSet[j].destPortleft_) && (filterSet[i].destPortright_ >= filterSet[j].destPortright_) ) &&
                                   ( (filterSet[i].srcPortleft_  <= filterSet[j].srcPortleft_)  && (filterSet[i].srcPortright_  >= filterSet[j].srcPortright_) ) &&
                                   ( (filterSet[i].pro_num_ == filterSet[j].pro_num_) || (filterSet[i].pro_num_ == 0) )
                                   ){}
                                else
                                    total++;
                            }
                        }
                        
                    }
                }
            }
        }
    }
    
}


int main(int argc, char *argv[])
{

    ofstream fout;
    double sort_start, hconflict_start, g1conflict_start, g2conflict_start, g0conflict_start;
    double sort_time, hconflict_time, g1conflict_time, g2conflict_time, g0conflict_time;
    int db_size;
    int total = 0;// for host computaion result
    int *d_total;// store device result
    int *h_total;// store host result
    int sum = 0;

    // CUDA variable
    filter* dev_filterSet;	// store filter in device
   
    //insert filters
    if (db.loadFilters(argv[1]) == -1)
	err_quit("Error: unable to open the filter database.");

    db_size = db.size();	// get database size
    printf("number of rules: %d\n", db_size);
    int block = 512;
    int grid = 4;
    //int grid =( db_size + block - 1)/block;
    //if(grid == 0)
	//grid = 1;
    // set to 0 for copying to device variable

    //hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    //hipSetDevice(0);

// ################ Prepare call M0 funtion ###################
    // prepare device variable
    hipMalloc((void**)&dev_filterSet, db_size * sizeof(filter));
    cudaCheckErrors("cudaMalloc1 fail");
    hipMalloc((void**)&d_total, grid*sizeof(int));
    cudaCheckErrors("cudaMalloc2 fail");   
    hipMemset(d_total, 0, grid*sizeof(int));

    h_total = (int*)malloc(grid*sizeof(int));
    // call kernel function
    g0conflict_start = cpuSecond();
    // copy to device
    hipMemcpy(dev_filterSet, db.filterSet_, db_size * sizeof(filter), hipMemcpyHostToDevice);
    cudaCheckErrors("cudaMemcpy1 fail");
    gpu_conflict_detect_0<<<grid, block>>>(dev_filterSet, db_size, d_total);
    hipDeviceSynchronize();
    
    // copy device result ot host
    hipMemcpy(h_total, d_total, grid*sizeof(int), hipMemcpyDeviceToHost);
    sum = 0;
    for(int tmp = 0; tmp < grid; tmp++)
	sum += h_total[tmp];
    
    g0conflict_time = cpuSecond() - g0conflict_start;
    cout << "m0_detection_time: " << g0conflict_time << " usec" << endl;
    printf("m0 conflict detection: %d\n", sum);
    free(h_total);
    hipFree(dev_filterSet);
    hipFree(d_total);


    // sort filter with srcIPlen & desIPlen
    sort_start = cpuSecond();
    bubble_sort(db, db_size);
    sort_time = cpuSecond() - sort_start;
    //cout << "sort_time: " << sort_time << " usec" << endl;


// ################ Prepare call M1 funtion ###################
    // prepare device variable
    hipMalloc((void**)&dev_filterSet, db_size * sizeof(filter));
    cudaCheckErrors("cudaMalloc1 fail");
    hipMalloc((void**)&d_total, grid*sizeof(int));
    cudaCheckErrors("cudaMalloc2 fail");
    hipMemset(d_total, 0, grid*sizeof(int));
    h_total = (int*)malloc(grid*sizeof(int));
    // call kernel function
    g1conflict_start = cpuSecond();
    // copy to device
    hipMemcpy(dev_filterSet, db.filterSet_, db_size * sizeof(filter), hipMemcpyHostToDevice);
    cudaCheckErrors("cudaMemcpy1 fail");
    gpu_conflict_detect_1<<<grid, block>>>(dev_filterSet, db_size, d_total);
    hipDeviceSynchronize();
    
    // copy device result ot host
    hipMemcpy(h_total, d_total, grid*sizeof(int), hipMemcpyDeviceToHost);
    sum = 0;
    for(int tmp = 0; tmp < grid; tmp++)
	sum += h_total[tmp];
     
    g1conflict_time = cpuSecond() - g1conflict_start;
    cout << "m1_detection_time: " << g1conflict_time << " usec" << endl;
    printf("m1 conflict detection: %d\n", sum);
    free(h_total);
    hipFree(dev_filterSet);
    hipFree(d_total);



// ################ Prepare call M2 funtion ###################
    // prepare device variable
    hipMalloc((void**)&dev_filterSet, db_size * sizeof(filter));
    cudaCheckErrors("cudaMalloc1 fail");
    hipMalloc((void**)&d_total, grid*sizeof(int));
    cudaCheckErrors("cudaMalloc2 fail");
    hipMemset(d_total, 0, grid*sizeof(int));
    h_total = (int*)malloc(grid*sizeof(int));
      
    // call kernel function method 2
    g2conflict_start = cpuSecond();
    // copy to device
    hipMemcpy(dev_filterSet, db.filterSet_, db_size * sizeof(filter), hipMemcpyHostToDevice);
    cudaCheckErrors("cudaMemcpy1 fail");
    gpu_conflict_detect_2<<<grid, block>>>(dev_filterSet, db_size, d_total);
    hipDeviceSynchronize();
    
    // copy device result ot host
    hipMemcpy(h_total, d_total, grid*sizeof(int), hipMemcpyDeviceToHost);
    sum = 0;
    for(int tmp = 0; tmp < grid; tmp++)
	sum += h_total[tmp];
    
    g2conflict_time = cpuSecond() - g2conflict_start;
    cout << "m2_detection_time: " << g2conflict_time << " usec" << endl;
    printf("m2 conflict detection: %d\n", sum);
    free(h_total);
    hipFree(dev_filterSet);
    hipFree(d_total);


    hipDeviceReset();
/*
// ################ Prepare call host funtion ###################
    hconflict_start = cpuSecond();
    host_conflict_detect(db.filterSet_, db_size, total);
    hconflict_time = cpuSecond() - hconflict_start;
    cout.setf(ios::fixed);
    cout << std::noshowpoint << "host_detection_time: " << hconflict_time << " usec" << endl;
    printf("Host conflict detection: %d\n", total);
*/    
/*
fout.open("0726_10248M1.txt", ios::app);   
//fout<< hconflict_time<< "\t";
fout<< g1conflict_time<< "\n";
//fout<< g1conflict_time<< "\t";
//fout<< g2conflict_time<< "\n";
fout.close();
*/
    //free(cpy_dev);
    //hipFree(dev_filterSet);
    //hipFree(dev_total);
}


